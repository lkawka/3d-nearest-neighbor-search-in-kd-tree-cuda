#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <time.h>

#define eChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

const int N = 5, DIM_SIZE = 3;

typedef struct __align__(16) {
    int3 value;
    int splitDim;
} KDNode;

void runAndTime(void (*f)());
void cpu();
void gpu();


int main() {
    srand(16);

    runAndTime(cpu);
    runAndTime(gpu);
}

void runAndTime(void (*f)())
{
    auto start = std::chrono::system_clock::now();
    f();
    auto end = std::chrono::system_clock::now();
    float duration = 1000.0 * std::chrono::duration<float>(end - start).count();
    std::cout << "Elapsed time in milliseconds : " << duration << "ms\n\n";
}

void generatePoints(int3 *points, int n) {
    for(int i = 0; i < n; i++) {
        points[i] = make_int3(rand()%100, rand()%100, rand()%100);
    }
}

void buildKdTree(int3 *points, KDNode *tree, int n) {
    for(int i = 0; i < n; i++) {
        tree[i] = { .value = points[i] };
    }
}

void cpu() {
    int3 *points = new int[N]];
    KDNode *tree = new KDNode[N];

    generatePoints(points, N);
    buildKdTree(points, tree, N);
}

void gpu()
{
    int3 *points;
    KDNode *tree;

    eChk(hipMallocManaged(&points, N * sizeof(int3)));
    eChk(hipMallocManaged(&tree, N * sizeof(KDNode)));

    generatePoints(points, N);
    buildKdTree(points, tree, N);

    eChk(hipFree(points));
}

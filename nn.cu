
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <time.h>
#include <algorithm>
#include <math.h>

#define eChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

const int N_POINTS = 1e6, N_QUERIES = 1e5, INF = 1e9;

void runAndTime(void (*f)(int3*, int, int3*, int), int3 *tree, int treeSize, int3 *queries, int nQueries);
void print(int3 *points, int n);
void generatePoints(int3 *points, int n);
void buildKDTree(int3 *points, int3 *tree, int n, int m);
void cpu(int3 *tree, int treeSize, int3 *queries, int nQueries);
void gpu(int3 *tree, int treeSize, int3 *queries, int nQueries);


int main() {
    srand(16);

    int TREE_SIZE = 1;
    while(TREE_SIZE < N_POINTS) TREE_SIZE <<= 1;

    int3 *points;
    int3 *tree;
    int3 *queries;
    eChk(hipMallocManaged(&points, N_POINTS * sizeof(int3)));
    eChk(hipMallocManaged(&tree, TREE_SIZE * sizeof(int3)));
    eChk(hipMallocManaged(&queries, N_QUERIES * sizeof(int3)));

    generatePoints(points, N_POINTS);
    buildKDTree(points, tree, N_POINTS, TREE_SIZE);
    generatePoints(queries, N_QUERIES);

    runAndTime(cpu, tree, TREE_SIZE, queries, N_QUERIES);
    runAndTime(gpu, tree, TREE_SIZE, queries, N_QUERIES);

    eChk(hipFree(points));
    eChk(hipFree(tree));
    eChk(hipFree(queries));
}

void runAndTime(void (*f)(int3*, int, int3*, int), int3 *tree, int treeSize, int3 *queries, int nQueries)
{
    auto start = std::chrono::system_clock::now();
    f(tree, treeSize, queries, nQueries);
    auto end = std::chrono::system_clock::now();
    float duration = 1000.0 * std::chrono::duration<float>(end - start).count();
    std::cout << "Elapsed time in milliseconds : " << duration << "ms\n\n";
}

void generatePoints(int3 *points, int n) {
    for(int i = 0; i < n; i++) {
        points[i] = make_int3(rand()%100, rand()%100, rand()%100);
    }
}


void buildSubTree(int3 *points, int3 *tree, int start, int end, int depth, int node) {
    if(start >= end) {
        return;
    }

    std::sort(points+start, points+end, [depth](int3 p1, int3 p2) -> bool {
        if(depth % 3 == 0) return p1.x < p2.x;
        if(depth % 3 == 1) return p1.y < p2.y;
        return p1.z < p2.z;
    });

    int split = (start + end-1)/2;

    tree[node].x = points[split].x;
    tree[node].y = points[split].y;
    tree[node].z = points[split].z;

    buildSubTree(points, tree, start, split, depth+1, node*2);
    buildSubTree(points, tree, split+1, end, depth+1, node*2 + 1);
}

void buildKDTree(int3 *points, int3 *tree, int n, int treeSize) {
    for(int i = 0; i < treeSize; i++) {
        tree[i] = make_int3(-INF, -INF, -INF);
    }

    buildSubTree(points, tree, 0, n, 0, 1);
}

void print(int3 *points, int n) {
    for(int i = 0; i < n; i++) {
        std::cout<<"["<<points[i].x<<", "<<points[i].y<<", "<<points[i].z<<"] ";
    }
    std::cout<<std::endl;
}

__device__ __host__ int3 closer(int3 p, int3 p2, int3 p3) {
    if((pow((double)(p.x-p2.x), 2)+pow((double)(p.y-p2.y), 2)+pow((double)(p.z-p2.z), 2)) < (pow((double)(p.x-p3.x), 2)+pow((double)(p.y-p3.y), 2)+pow((double)(p.z-p3.z), 2))) {
        return p2;
    }
    return p3;
}

__device__ __host__ int3 findNearestNeighbor(int3 *tree, int treeSize, int treeNode, int depth, int3 query) {
    int3 result = tree[treeNode];

    int val1, val2;
    if(depth % 3 == 0) {
        val1 = result.x;
        val2 = query.x;
    } else if(depth % 3 == 1) {
        val1 = result.y;
        val2 = query.y;
    } else {
        val1 = result.z;
        val2 = query.z;
    }

    if(val1 < val2) {
        if(treeNode*2 < treeSize && tree[treeSize*2].x != -INF && tree[treeSize*2].y != -INF && tree[treeSize*2].z != -INF) {
            return closer(query, result, findNearestNeighbor(tree, treeSize, treeNode*2, depth+1, query));
        }
    } else if(val1 > val2) {
        if(treeNode*2+1 < treeSize && tree[treeSize*2+1].x != -INF && tree[treeSize*2+1].y != -INF && tree[treeSize*2+1].z != -INF) {
            return closer(query, result, findNearestNeighbor(tree, treeSize, treeNode*2+1, depth+1, query));
        }
    }
    return result;
}

void cpu(int3 *tree, int treeSize, int3 *queries, int nQueries) {
    int3 *results = new int3[nQueries];

    for(int i = 0; i < nQueries; i++) {
        results[i] = findNearestNeighbor(tree, treeSize, 1, 0, queries[i]);
    }

    // print(queries, nQueries);
    // print(results, nQueries);
    std::cout<<"CPU done\n";
}

__global__ void nearestNeighborGPU(int3 *tree, int treeSize, int3 *queries, int3 *results, int nQueries) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < nQueries) {
        results[index] = findNearestNeighbor(tree, treeSize, 1, 0, queries[index]);
    }
}

void gpu(int3 *tree, int treeSize, int3 *queries, int nQueries)
{
    int3 *results;
    eChk(hipMallocManaged(&results, nQueries * sizeof(int3)));

    nearestNeighborGPU<<<1, 256>>>(tree, treeSize, queries, results, nQueries);

    eChk(hipDeviceSynchronize());

    // print(results, nQueries);
    eChk(hipFree(results));
}
